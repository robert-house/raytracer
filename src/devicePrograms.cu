#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "./renderer/ConstantBuffer.h"
#include "math.h"
#include "random.h"

using namespace gdt;

struct PerRayData
{
    vec3f       direction = vec3f(0.0f);
    vec3f       origin    = vec3f(0.0f);
    vec3f       color     = vec3f(0.0f);
    vec3f       emitted   = vec3f(0.0f);
    bool        done      = false;
};

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ ConstantBuffer optixLaunchParams;

// for this simple example, we have a single ray type
enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

static __forceinline__ __device__
void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ vec3f reflectRay(const vec3f& rayDir, const vec3f& normal)
{
    //return (rayDir - 2.0f * dot(rayDir / length(rayDir), normal) * normal) + .0f * rnd(prd.seed);
    return (rayDir - 2.0f * dot(rayDir / length(rayDir), normal) * normal);
}

static __forceinline__ __device__ vec3f calculateNormals(const TriangleMeshSBTData& sbtData, const vec3i& index, const vec2f& uv)
{
    // Verts
    const vec3f& v0 = sbtData.vertex[index.x];
    const vec3f& v1 = sbtData.vertex[index.y];
    const vec3f& v2 = sbtData.vertex[index.z];
    
    vec3f N;
    
    if (sbtData.normals)
    {
        N = (1.0f - uv.u - uv.v)
            * sbtData.normals[index.x]
            + uv.u * sbtData.normals[index.y]
            + uv.v * sbtData.normals[index.z];
    }
    else
    {
        N = normalize(cross(v1 - v0, v2 - v0));
    }

    N = normalize(N);

    return N;
}

static __forceinline__ __device__ vec3f mapTextures(const TriangleMeshSBTData& sbtData, const vec3i& index, const vec2f& uv)
{
    // Diffuse color and textures
    vec3f diffuseColor = sbtData.color;

    if (sbtData.hasTexture && sbtData.texcoords)
    {
        const vec2f tc = (1.f - uv.u - uv.v)
            * sbtData.texcoords[index.x]
            + uv.u * sbtData.texcoords[index.y]
            + uv.v * sbtData.texcoords[index.z];

        vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor *= (vec3f)fromTexture;
    }

    return diffuseColor;
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__radiance()
{
    const TriangleMeshSBTData& sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    PerRayData& prd = *(PerRayData*)getPRD<vec3f>();

    // Get Primitive Index Information
    const int   primitiveIndex = optixGetPrimitiveIndex();
    const vec3i index = sbtData.index[primitiveIndex];

    // Get UVs for this Primitive
    const vec2f uv(optixGetTriangleBarycentrics().x, optixGetTriangleBarycentrics().y);

    // Compute Interpolated Per-Vertex Normals with Per-Face normals as fallback
    vec3f N = calculateNormals(sbtData, index, uv);

    // Diffuse color and textures
    vec3f diffuseColor = mapTextures(sbtData, index, uv);

    // Get ray direction and origin from Optix
    const vec3f rayDir = optixGetWorldRayDirection();
    const vec3f origin = optixGetWorldRayOrigin();

    // Compute next origin (point of impact) and store it in our
    // per-ray data struct for the next trace
    const vec3f P = origin + (optixGetRayTmax() * rayDir);
    prd.origin = P;

    // Compute the new direction and radiance transfer based
    // of of the material on this primitive
    const float cosDN = 0.2f + .8f * fabsf(dot(rayDir, N)); // phong lighting?
    prd.color = cosDN * diffuseColor;
    prd.done = true; // Finished Tracing

    // Saved for metal material
    //prd.direction = reflectRay(rayDir, N);
}

extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{
    PerRayData& prd = *(PerRayData*)getPRD<vec3f>();

    // Set to constant black as background color and mark that
    // we should stop tracing this secondary ray
    prd.color = vec3f(0.0f);
    prd.done = true;
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    // This is our perRay data structure similar to HitRecord
    // in software mode
    PerRayData perRayData;
    
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    // Because we are accumulating, init finalColor to 1.0f
    vec3f finalColor(0.0f);

    const auto& camera = optixLaunchParams.camera;

    // normalized screen plane position, in [0,1]^2
    const vec2f screen(vec2f(ix + .5f, iy + .5f)
        / vec2f(optixLaunchParams.frame.size));

    vec3f rayOrigin = camera.position;

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&perRayData, u0, u1);

    // generate ray direction
    vec3f rayDirection = normalize(camera.direction
        + (screen.x - 0.5f) * camera.horizontal
        + (screen.y - 0.5f) * camera.vertical);

    // Quick Hack for multi sampling
    int numberOfSamples = 10;
    int depth = 0;
    int maxBounces = 3;

    // While we are not done yet, spawn rays!
    while (!perRayData.done && depth < maxBounces)
    {
        optixTrace(optixLaunchParams.traversable,
            rayOrigin,
            rayDirection,
            0.01f,    // tmin
            1e16f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
            SURFACE_RAY_TYPE,             // SBT offset
            RAY_TYPE_COUNT,               // SBT stride
            SURFACE_RAY_TYPE,             // missSBTIndex 
            u0, u1);

        // Want to accumulate the color here per trace
        finalColor += perRayData.color;

        // Set new ray direction and origin from previous ray intersection
        rayOrigin    = perRayData.origin;
        rayDirection = perRayData.direction;

        // Increase the depth, or number of secondary relections allowed before
        // we have to cut off the trace
        depth++;
    }

    // Write RGB Values
    const int r = int(255.99f * finalColor.x);
    const int g = int(255.99f * finalColor.y);
    const int b = int(255.99f * finalColor.z);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
        | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}